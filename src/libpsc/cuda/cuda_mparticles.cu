#include "hip/hip_runtime.h"

#include "cuda_mparticles.h"
#include "cuda_bits.h"

#include "psc_bits.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include <cstdio>
#include <cassert>

// ----------------------------------------------------------------------
// ctor

cuda_mparticles::cuda_mparticles(mrc_json_t json)
{
  std::memset(this, 0, sizeof(*this)); // FIXME

  mrc_json_t json_info = mrc_json_get_object_entry(json, "info");
  
  n_patches = mrc_json_get_object_entry_integer(json_info, "n_patches");
  mrc_json_get_object_entry_int3(json_info, "ldims", ldims);
  mrc_json_get_object_entry_int3(json_info, "bs", bs);
  double dx[3];
  mrc_json_get_object_entry_double3(json_info, "dx", dx);

  for (int d = 0; d < 3; d++) {
    dx[d] = dx[d];
    assert(ldims[d] % bs[d] == 0);
    b_mx[d] = ldims[d] / bs[d];
    b_dxi[d] = 1.f / (bs[d] * dx[d]);
  }
  
  xb_by_patch = new float_3[n_patches];
  mrc_json_t json_xb_by_patch = mrc_json_get_object_entry(json_info, "xb_by_patch");
  for (int p = 0; p < n_patches; p++) {
    mrc_json_get_float3(mrc_json_get_array_entry(json_xb_by_patch, p), xb_by_patch[p]);
  }

  fnqs = mrc_json_get_object_entry_double(json_info, "fnqs");
  eta  = mrc_json_get_object_entry_double(json_info, "eta");
  dt   = mrc_json_get_object_entry_double(json_info, "dt");

  mrc_json_t json_kind_q = mrc_json_get_object_entry(json_info, "kind_q");
  n_kinds = mrc_json_get_array_length(json_kind_q);
  kind_q = new float[n_kinds];
  // FIXME, could use a mrc_json helper
  for (int k = 0; k < n_kinds; k++) {
    kind_q[k] = mrc_json_get_array_entry_double(json_kind_q, k);
  }
  mrc_json_t json_kind_m = mrc_json_get_object_entry(json_info, "kind_m");
  assert(n_kinds == mrc_json_get_array_length(json_kind_m));
  kind_m = new float[n_kinds];
  // FIXME, could use a mrc_json helper
  for (int k = 0; k < n_kinds; k++) {
    kind_m[k] = mrc_json_get_array_entry_double(json_kind_m, k);
  }

  n_blocks_per_patch = b_mx[0] * b_mx[1] * b_mx[2];
  n_blocks = n_patches * n_blocks_per_patch;

  hipError_t ierr;

  ierr = hipMalloc(&d_off, (n_blocks + 1) * sizeof(*d_off)); cudaCheck(ierr);
  ierr = hipMemset(d_off, 0, (n_blocks + 1) * sizeof(*d_off)); cudaCheck(ierr);

  cuda_mparticles_bnd_setup(this);
}

// ----------------------------------------------------------------------
// cuda_mparticles_free_particle_mem

static void
cuda_mparticles_free_particle_mem(struct cuda_mparticles *cmprts)
{
  hipError_t ierr;

  ierr = hipFree(cmprts->d_xi4); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_pxi4); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_alt_xi4); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_alt_pxi4); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_bidx); cudaCheck(ierr);
  ierr = hipFree(cmprts->d_id); cudaCheck(ierr);

  cuda_mparticles_bnd_free_particle_mem(cmprts);
}

// ----------------------------------------------------------------------
// dtor

cuda_mparticles::~cuda_mparticles()
{
  hipError_t ierr;

  ierr = hipFree(d_off); cudaCheck(ierr);

  cuda_mparticles_free_particle_mem(this);
  cuda_mparticles_bnd_destroy(this);
  
  delete[] xb_by_patch;
  delete[] kind_q;
  delete[] kind_m;
}

// ----------------------------------------------------------------------
// cuda_mparticles_reserve_all

void
cuda_mparticles_reserve_all(struct cuda_mparticles *cmprts, unsigned int *n_prts_by_patch)
{
  hipError_t ierr;

  unsigned int size = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    size += n_prts_by_patch[p];
  }

  if (size <= cmprts->n_alloced) {
    return;
  }

  size *= 1.2;// FIXME hack
  unsigned int n_alloced = max(size, 2 * cmprts->n_alloced);

  if (cmprts->n_alloced > 0) {
    cuda_mparticles_free_particle_mem(cmprts);
  }
  cmprts->n_alloced = n_alloced;

  ierr = hipMalloc((void **) &cmprts->d_xi4, n_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_pxi4, n_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_alt_xi4, n_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_alt_pxi4, n_alloced * sizeof(float4)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_bidx, n_alloced * sizeof(unsigned int)); cudaCheck(ierr);
  ierr = hipMalloc((void **) &cmprts->d_id, n_alloced * sizeof(unsigned int)); cudaCheck(ierr);

  cuda_mparticles_bnd_reserve_all(cmprts);
}

// ----------------------------------------------------------------------
// cuda_mparticles_to_device

void
cuda_mparticles_to_device(struct cuda_mparticles *cmprts, float_4 *xi4, float_4 *pxi4,
			  unsigned int n_prts, unsigned int off)
{
  hipError_t ierr;

  assert(off + n_prts <= cmprts->n_alloced);
  ierr = hipMemcpy(cmprts->d_xi4 + off, xi4, n_prts * sizeof(*xi4),
		    hipMemcpyHostToDevice); cudaCheck(ierr);
  ierr = hipMemcpy(cmprts->d_pxi4 + off, pxi4, n_prts * sizeof(*pxi4),
		    hipMemcpyHostToDevice); cudaCheck(ierr);
}

// ----------------------------------------------------------------------
// cuda_mparticles_from_device

void
cuda_mparticles_from_device(struct cuda_mparticles *cmprts, float_4 *xi4, float_4 *pxi4,
			    unsigned int n_prts, unsigned int off)
{
  hipError_t ierr;

  assert(off + n_prts <= cmprts->n_alloced);
  ierr = hipMemcpy(xi4, cmprts->d_xi4 + off, n_prts * sizeof(*xi4),
		    hipMemcpyDeviceToHost); cudaCheck(ierr);
  ierr = hipMemcpy(pxi4, cmprts->d_pxi4 + off, n_prts * sizeof(*pxi4),
		    hipMemcpyDeviceToHost); cudaCheck(ierr);
}

// ----------------------------------------------------------------------
// cuda_mparticles_dump_by_patch

void
cuda_mparticles_dump_by_patch(struct cuda_mparticles *cmprts, unsigned int *n_prts_by_patch)
{
  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  thrust::device_ptr<float4> d_pxi4(cmprts->d_pxi4);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);

  printf("cuda_mparticles_dump_by_patch: n_prts = %d\n", cmprts->n_prts);
  unsigned int off = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    float *xb = &cmprts->xb_by_patch[p][0];
    for (int n = 0; n < n_prts_by_patch[p]; n++) {
      float4 xi4 = d_xi4[n + off], pxi4 = d_pxi4[n + off];
      unsigned int bidx = d_bidx[n + off], id = d_id[n + off];
      printf("cuda_mparticles_dump_by_patch: [%d/%d] %g %g %g // %d // %g %g %g // %g b_idx %d id %d\n",
	     p, n, xi4.x + xb[0], xi4.y + xb[1], xi4.z + xb[2],
	     cuda_float_as_int(xi4.w),
	     pxi4.x, pxi4.y, pxi4.z, pxi4.w,
	     bidx, id);
    }
    off += n_prts_by_patch[p];
  }
}

// ----------------------------------------------------------------------
// cuda_mparticles_dump

void
cuda_mparticles_dump(struct cuda_mparticles *cmprts)
{
  int n_prts = cmprts->n_prts;
  
  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  thrust::device_ptr<float4> d_pxi4(cmprts->d_pxi4);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);

  printf("cuda_mparticles_dump: n_prts = %d\n", n_prts);
  unsigned int off = 0;
  for (int b = 0; b < cmprts->n_blocks; b++) {
    unsigned int off_b = d_off[b], off_e = d_off[b+1];
    int p = b / cmprts->n_blocks_per_patch;
    printf("cuda_mparticles_dump: block %d: %d -> %d (patch %d)\n", b, off_b, off_e, p);
    assert(d_off[b] == off);
    for (int n = d_off[b]; n < d_off[b+1]; n++) {
      float4 xi4 = d_xi4[n], pxi4 = d_pxi4[n];
      unsigned int bidx = d_bidx[n], id = d_id[n];
      printf("cuda_mparticles_dump: [%d] %g %g %g // %d // %g %g %g // %g || bidx %d id %d\n",
	     n, xi4.x, xi4.y, xi4.z, cuda_float_as_int(xi4.w), pxi4.x, pxi4.y, pxi4.z, pxi4.w,
	     bidx, id);
      assert(b == bidx);
    }
    off += off_e - off_b;
  }
}

// ----------------------------------------------------------------------
// cuda_mparticles_swap_alt

void
cuda_mparticles_swap_alt(struct cuda_mparticles *cmprts)
{
  float4 *tmp_xi4 = cmprts->d_alt_xi4;
  float4 *tmp_pxi4 = cmprts->d_alt_pxi4;
  cmprts->d_alt_xi4 = cmprts->d_xi4;
  cmprts->d_alt_pxi4 = cmprts->d_pxi4;
  cmprts->d_xi4 = tmp_xi4;
  cmprts->d_pxi4 = tmp_pxi4;
}

// ----------------------------------------------------------------------
// cuda_params2

struct cuda_params2 {
  unsigned int b_mx[3];
  float b_dxi[3];
};

static void
cuda_params2_set(struct cuda_params2 *prm, const struct cuda_mparticles *cuda_mprts)
{
  for (int d = 0; d < 3; d++) {
    prm->b_mx[d]  = cuda_mprts->b_mx[d];
    prm->b_dxi[d] = cuda_mprts->b_dxi[d];
  }
}

static void
cuda_params2_free(struct cuda_params2 *prm)
{
}

#define THREADS_PER_BLOCK 256

// ----------------------------------------------------------------------
// get_block_idx

static int
get_block_idx(struct cuda_mparticles *cmprts, float4 xi4, int p)
{
  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  float *b_dxi = cmprts->b_dxi;
  int *b_mx = cmprts->b_mx;
  
  unsigned int block_pos_y = (int) floorf(xi4.y * b_dxi[1]);
  unsigned int block_pos_z = (int) floorf(xi4.z * b_dxi[2]);

  int bidx;
  if (block_pos_y >= b_mx[1] || block_pos_z >= b_mx[2]) {
    bidx = -1;
  } else {
    bidx = (p * b_mx[2] + block_pos_z) * b_mx[1] + block_pos_y;
  }

  return bidx;
}

// ----------------------------------------------------------------------
// cuda_mprts_find_block_indices_ids

__global__ static void
mprts_find_block_indices_ids(struct cuda_params2 prm, float4 *d_xi4, unsigned int *d_off,
			     unsigned int *d_bidx, unsigned int *d_ids, int n_patches,
			     int n_blocks_per_patch)
{
  int n = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
  int nr_blocks = prm.b_mx[1] * prm.b_mx[2];

  for (int p = 0; p < n_patches; p++) {
    unsigned int off = d_off[p * n_blocks_per_patch];
    unsigned int n_prts = d_off[(p + 1) * n_blocks_per_patch] - off;
    if (n < n_prts) {
      float4 xi4 = d_xi4[n + off];
      unsigned int block_pos_y = __float2int_rd(xi4.y * prm.b_dxi[1]);
      unsigned int block_pos_z = __float2int_rd(xi4.z * prm.b_dxi[2]);
      
      int block_idx;
      if (block_pos_y >= prm.b_mx[1] || block_pos_z >= prm.b_mx[2]) {
	block_idx = -1; // not supposed to happen here!
      } else {
	block_idx = block_pos_z * prm.b_mx[1] + block_pos_y + p * nr_blocks;
      }
      d_bidx[n + off] = block_idx;
      d_ids[n + off] = n + off;
    }
  }
}

void
cuda_mparticles_find_block_indices_ids(struct cuda_mparticles *cmprts)
{
  if (cmprts->n_patches == 0) {
    return;
  }

  // OPT: if we didn't need max_n_prts, we wouldn't have to get the
  // sizes / offsets at all, and it seems likely we could do a better
  // job here in general
  unsigned int n_prts_by_patch[cmprts->n_patches];
  cuda_mparticles_get_size_all(cmprts, n_prts_by_patch);
  
  int max_n_prts = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    if (n_prts_by_patch[p] > max_n_prts) {
      max_n_prts = n_prts_by_patch[p];
    }
  }

  struct cuda_params2 prm;
  cuda_params2_set(&prm, cmprts);
    
  dim3 dimGrid((max_n_prts + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
  dim3 dimBlock(THREADS_PER_BLOCK);

  mprts_find_block_indices_ids<<<dimGrid, dimBlock>>>(prm,
						      cmprts->d_xi4, 
						      cmprts->d_off,
						      cmprts->d_bidx,
						      cmprts->d_id,
						      cmprts->n_patches,
						      cmprts->n_blocks_per_patch);
  cuda_sync_if_enabled();
  cuda_params2_free(&prm);
}

// ----------------------------------------------------------------------
// cuda_mparticles_reorder_and_offsets

__global__ static void
mprts_reorder_and_offsets(int nr_prts, float4 *xi4, float4 *pxi4, float4 *alt_xi4, float4 *alt_pxi4,
			  unsigned int *d_bidx, unsigned int *d_ids, unsigned int *d_off, int last_block)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i > nr_prts)
    return;

  int block, prev_block;
  if (i < nr_prts) {
    alt_xi4[i] = xi4[d_ids[i]];
    alt_pxi4[i] = pxi4[d_ids[i]];
    
    block = d_bidx[i];
  } else { // needed if there is no particle in the last block
    block = last_block;
  }

  // OPT: d_bidx[i-1] could use shmem
  // create offsets per block into particle array
  prev_block = -1;
  if (i > 0) {
    prev_block = d_bidx[i-1];
  }
  for (int b = prev_block + 1; b <= block; b++) {
    d_off[b] = i;
  }
}

void
cuda_mparticles_reorder_and_offsets(struct cuda_mparticles *cmprts)
{
  if (cmprts->n_patches == 0) {
    return;
  }

  dim3 dimGrid((cmprts->n_prts + 1 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
  dim3 dimBlock(THREADS_PER_BLOCK);

  mprts_reorder_and_offsets<<<dimGrid, dimBlock>>>(cmprts->n_prts, cmprts->d_xi4, cmprts->d_pxi4,
						   cmprts->d_alt_xi4, cmprts->d_alt_pxi4,
						   cmprts->d_bidx, cmprts->d_id,
						   cmprts->d_off, cmprts->n_blocks);
  cuda_sync_if_enabled();

  cuda_mparticles_swap_alt(cmprts);
  cmprts->need_reorder = false;
}

void
cuda_mparticles_reorder_and_offsets_slow(struct cuda_mparticles *cmprts)
{
  if (cmprts->n_patches == 0) {
    return;
  }

  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  thrust::device_ptr<float4> d_pxi4(cmprts->d_pxi4);
  thrust::device_ptr<float4> d_alt_xi4(cmprts->d_alt_xi4);
  thrust::device_ptr<float4> d_alt_pxi4(cmprts->d_alt_pxi4);
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);

  thrust::host_vector<float4> h_xi4(d_xi4, d_xi4 + cmprts->n_prts);
  thrust::host_vector<float4> h_pxi4(d_pxi4, d_pxi4 + cmprts->n_prts);
  thrust::host_vector<float4> h_alt_xi4(d_alt_xi4, d_alt_xi4 + cmprts->n_prts);
  thrust::host_vector<float4> h_alt_pxi4(d_alt_pxi4, d_alt_pxi4 + cmprts->n_prts);
  thrust::host_vector<unsigned int> h_off(d_off, d_off + cmprts->n_blocks + 1);
  thrust::host_vector<unsigned int> h_bidx(d_bidx, d_bidx + cmprts->n_prts);
  thrust::host_vector<unsigned int> h_id(d_id, d_id + cmprts->n_prts);

  for (int i = 0; i <= cmprts->n_prts; i++) {
    //    unsigned int bidx;
    unsigned int block;
    if (i < cmprts->n_prts) {
      h_alt_xi4[i] = h_xi4[h_id[i]];
      h_alt_pxi4[i] = h_pxi4[h_id[i]];
      //bidx = get_block_idx(cmprts, h_alt_xi4[i], 0);
      block = h_bidx[i];
    } else {
      //bidx = cmprts->n_blocks;
      block = cmprts->n_blocks;
    }
    // if (i < 10) {
    //   printf("i %d bidx %d block %d xi4 %g %g\n", bidx, block, h_alt_xi4[i].y, h_alt_xi4[i].z);
    // }
    int prev_block = (i > 0) ? (int) h_bidx[i-1] : -1;
    for (int b = prev_block + 1; b <= block; b++) {
      h_off[b] = i;
    }
  }

  thrust::copy(h_alt_xi4.begin(), h_alt_xi4.end(), d_alt_xi4);
  thrust::copy(h_alt_pxi4.begin(), h_alt_pxi4.end(), d_alt_pxi4);
  thrust::copy(h_off.begin(), h_off.end(), d_off);
  
  cuda_mparticles_swap_alt(cmprts);
  cmprts->need_reorder = false;
}

// ----------------------------------------------------------------------
// cuda_mparticles_check_in_patch_unordered_slow

void
cuda_mparticles_check_in_patch_unordered_slow(struct cuda_mparticles *cmprts,
					      unsigned int *nr_prts_by_patch)
{
  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);

  unsigned int off = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    for (int n = 0; n < nr_prts_by_patch[p]; n++) {
      int bidx = get_block_idx(cmprts, d_xi4[off + n], p);
      assert(bidx >= 0 && bidx <= cmprts->n_blocks);
    }
    off += nr_prts_by_patch[p];
  }

  assert(off == cmprts->n_prts);
  printf("PASS: cuda_mparticles_check_in_patch_unordered_slow()\n");
}

// ----------------------------------------------------------------------
// cuda_mparticles_check_bix_id_unordered_slow

void
cuda_mparticles_check_bidx_id_unordered_slow(struct cuda_mparticles *cmprts,
					     unsigned int *n_prts_by_patch)
{
  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);

  unsigned int off = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    for (int n = 0; n < n_prts_by_patch[p]; n++) {
      int bidx = get_block_idx(cmprts, d_xi4[off + n], p);
      assert(bidx == d_bidx[off+n]);
      assert(off+n == d_id[off+n]);
    }
    off += n_prts_by_patch[p];
  }

  assert(off == cmprts->n_prts);
  printf("PASS: cuda_mparticles_check_bidx_id_unordered_slow()\n");
}

// ----------------------------------------------------------------------
// cuda_mparticles_check_ordered_slow

void
cuda_mparticles_check_ordered_slow(struct cuda_mparticles *cmprts)
{
  bool need_reorder = cmprts->need_reorder;
  
  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);

  unsigned int off = 0;
  for (int b = 0; b < cmprts->n_blocks; b++) {
    int p = b / cmprts->n_blocks_per_patch;
    unsigned int off_b = d_off[b], off_e = d_off[b+1];
    assert(off_e >= off_b);
    // printf("cuda_mparticles_check_ordered: block %d: %d -> %d (patch %d)\n", b, off_b, off_e, p);
    assert(d_off[b] == off);
    for (int n = d_off[b]; n < d_off[b+1]; n++) {
      float4 xi4;
      if (need_reorder) {
	xi4 = d_xi4[d_id[n]];
      } else {
	xi4 = d_xi4[n];
      }
      unsigned int bidx = get_block_idx(cmprts, xi4, p);
      //printf("cuda_mparticles_check_ordered: bidx %d\n", bidx);
      if (b != bidx) {
	printf("b %d bidx %d n %d p %d xi4 %g %g %g\n",
	       b, bidx, n, p, xi4.x, xi4.y, xi4.z);
	unsigned int block_pos_y = (int) floorf(xi4.y * cmprts->b_dxi[1]);
	unsigned int block_pos_z = (int) floorf(xi4.z * cmprts->b_dxi[2]);
	printf("block_pos %d %d %g %g\n", block_pos_y, block_pos_z, xi4.y * cmprts->b_dxi[1],
	       xi4.z * cmprts->b_dxi[2]);
      }
      assert(b == bidx);
    }
    off += off_e - off_b;
  }
  assert(off == cmprts->n_prts);
  printf("cuda_mparticles_check_ordered: PASS\n");
}

// ----------------------------------------------------------------------
// cuda_mparticles_check_ordered

void
cuda_mparticles_check_ordered(struct cuda_mparticles *cmprts)
{
  bool need_reorder = cmprts->need_reorder;

  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::host_vector<float4> h_xi4(d_xi4, d_xi4 + cmprts->n_prts);
  thrust::host_vector<unsigned int> h_off(d_off, d_off + cmprts->n_blocks + 1);
  thrust::host_vector<unsigned int> h_id(d_id, d_id + cmprts->n_prts);

  //printf("cuda_mparticles_check_ordered: need_reorder %s\n", need_reorder ? "true" : "false");

  // for (int n = 0; n < 10; n++) {
  //   unsigned int bidx = d_bidx[n];
  //   printf("n %d bidx %d xi4 %g %g\n", n, bidx, h_xi4[n].y, h_xi4[n].z);
  // }
  unsigned int off = 0;
  for (int b = 0; b < cmprts->n_blocks; b++) {
    int p = b / cmprts->n_blocks_per_patch;
    unsigned int off_b = h_off[b], off_e = h_off[b+1];
    assert(off_e >= off_b);
    //printf("cuda_mparticles_check_ordered: block %d: %d -> %d (patch %d)\n", b, off_b, off_e, p);
    assert(off_b == off);
    for (int n = h_off[b]; n < h_off[b+1]; n++) {
      float4 xi4;
      if (need_reorder) {
	xi4 = h_xi4[h_id[n]];
      } else {
	xi4 = h_xi4[n];
      }
      unsigned int bidx = get_block_idx(cmprts, xi4, p);
      //printf("cuda_mparticles_check_ordered: bidx %d\n", bidx);
      if (b != bidx) {
	printf("b %d bidx %d n %d p %d xi4 %g %g %g\n",
	       b, bidx, n, p, xi4.x, xi4.y, xi4.z);
	unsigned int block_pos_y = (int) floorf(xi4.y * cmprts->b_dxi[1]);
	unsigned int block_pos_z = (int) floorf(xi4.z * cmprts->b_dxi[2]);
	printf("block_pos %d %d %g %g\n", block_pos_y, block_pos_z, xi4.y * cmprts->b_dxi[1],
	       xi4.z * cmprts->b_dxi[2]);
      }
      assert(b == bidx);
    }
    off += off_e - off_b;
  }
  assert(off == cmprts->n_prts);
  printf("cuda_mparticles_check_ordered: PASS\n");
}

// ----------------------------------------------------------------------
// cuda_mparticles_sort_initial

void
cuda_mparticles_sort_initial(struct cuda_mparticles *cmprts,
			     unsigned int *n_prts_by_patch)
{
}

// ----------------------------------------------------------------------
// cuda_mparticles_setup_internals

void
cuda_mparticles_setup_internals(struct cuda_mparticles *cmprts)
{
  static int first_time = false;
  if (first_time) {
    unsigned int n_prts_by_patch[cmprts->n_patches];
    cuda_mparticles_get_size_all(cmprts, n_prts_by_patch);
    cuda_mparticles_check_in_patch_unordered_slow(cmprts, n_prts_by_patch);
  }

  cuda_mparticles_find_block_indices_ids(cmprts);
  if (first_time) {
    unsigned int n_prts_by_patch[cmprts->n_patches];
    cuda_mparticles_get_size_all(cmprts, n_prts_by_patch);
    cuda_mparticles_check_bidx_id_unordered_slow(cmprts, n_prts_by_patch);
  }

  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);
  thrust::stable_sort_by_key(d_bidx, d_bidx + cmprts->n_prts, d_id);
  cuda_mparticles_reorder_and_offsets(cmprts);

  if (first_time) {
    cuda_mparticles_check_ordered(cmprts);
    first_time = false;
  }
}

// ----------------------------------------------------------------------
// cuda_mparticles_resize_all
//
// FIXME, this function currently is used in two contexts:
// - to implement mprts::resize_all(), but in this case we
//   need to be careful. It's destructive, which is unexpected.
//   we might want to only support (and check for) the case of
//   resizing from 0 size.
//   in this case, we also should check that things fit into what's
//   alloced (also: a very similar issues is cuda_mparticles_reserve_all()
//   which doesn't realloc but destroy, again that's unexpected behavior
// - to reset the internal n_prts_by_patch as part of sorting etc.
//   in that case, we supposedly know what we're doing, so we at most need
//   to check that we aren't beyond our allocated space

void
cuda_mparticles_resize_all(struct cuda_mparticles *cmprts,
			   const unsigned int *n_prts_by_patch)
{
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);
  thrust::host_vector<unsigned int> h_off(cmprts->n_blocks + 1);

  unsigned int off = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    h_off[p * cmprts->n_blocks_per_patch] = off;
    off += n_prts_by_patch[p];
    // printf("set_n_prts p%d: %d\n", p, n_prts_by_patch[p]);
  }
  h_off[cmprts->n_blocks] = off;
  cmprts->n_prts = off;

  thrust::copy(h_off.begin(), h_off.end(), d_off);
}

// ----------------------------------------------------------------------
// cuda_mparticles_get_n_prts

unsigned int
cuda_mparticles_get_n_prts(struct cuda_mparticles *cmprts)
{
  return cmprts->n_prts;
}

// ----------------------------------------------------------------------
// cuda_mparticles_get_size_all

void
cuda_mparticles_get_size_all(struct cuda_mparticles *cmprts,
			     unsigned int *n_prts_by_patch)
{
  thrust::device_ptr<unsigned int> d_off(cmprts->d_off);
  thrust::host_vector<unsigned int> h_off(d_off, d_off + cmprts->n_blocks + 1);

  for (int p = 0; p < cmprts->n_patches; p++) {
    n_prts_by_patch[p] = h_off[(p+1) * cmprts->n_blocks_per_patch] - h_off[p * cmprts->n_blocks_per_patch];
    //printf("p %d n_prts_by_patch %d\n", p, n_prts_by_patch[p]);
  }
}

// ----------------------------------------------------------------------
// cuda_mparticles_reorder

__global__ static void
k_cuda_mparticles_reorder(int nr_prts, unsigned int *d_ids,
		 float4 *xi4, float4 *pxi4,
		 float4 *alt_xi4, float4 *alt_pxi4)
{
  int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  if (i < nr_prts) {
    int j = d_ids[i];
    alt_xi4[i] = xi4[j];
    alt_pxi4[i] = pxi4[j];
  }
}

void
cuda_mparticles_reorder(struct cuda_mparticles *cmprts)
{
  if (!cmprts->need_reorder) {
    return;
  }
  
  dim3 dimGrid((cmprts->n_prts + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
  
  k_cuda_mparticles_reorder<<<dimGrid, THREADS_PER_BLOCK>>>
    (cmprts->n_prts, cmprts->d_id,
     cmprts->d_xi4, cmprts->d_pxi4,
     cmprts->d_alt_xi4, cmprts->d_alt_pxi4);
  
  cuda_mparticles_swap_alt(cmprts);

  cmprts->need_reorder = false;
}

// ----------------------------------------------------------------------
// cuda_mparticles_inject

void
cuda_mparticles_inject(struct cuda_mparticles *cmprts, struct cuda_mparticles_prt *buf,
		       unsigned int *buf_n_by_patch)
{
  if (cmprts->need_reorder) {
    cuda_mparticles_reorder(cmprts);
    cmprts->need_reorder = false;
  }
  
  unsigned int buf_n = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    buf_n += buf_n_by_patch[p];
    //    printf("p %d buf_n_by_patch %d\n", p, buf_n_by_patch[p]);
  }
  //  printf("buf_n %d\n", buf_n);

  thrust::host_vector<float4> h_xi4(buf_n);
  thrust::host_vector<float4> h_pxi4(buf_n);
  thrust::host_vector<unsigned int> h_bidx(buf_n);
  thrust::host_vector<unsigned int> h_id(buf_n);

  unsigned int off = 0;
  for (int p = 0; p < cmprts->n_patches; p++) {
    for (int n = 0; n < buf_n_by_patch[p]; n++) {
      float4 *xi4 = &h_xi4[off + n];
      float4 *pxi4 = &h_pxi4[off + n];
      cuda_mparticles_prt *prt = &buf[off + n];
      
      xi4->x  = prt->xi[0];
      xi4->y  = prt->xi[1];
      xi4->z  = prt->xi[2];
      xi4->w  = cuda_int_as_float(prt->kind);
      pxi4->x = prt->pxi[0];
      pxi4->y = prt->pxi[1];
      pxi4->z = prt->pxi[2];
      pxi4->w = prt->qni_wni;

      h_bidx[off + n] = get_block_idx(cmprts, *xi4, p);
      h_id[off + n] = cmprts->n_prts + off + n;
    }
    off += buf_n_by_patch[p];
  }
  assert(off == buf_n);

  unsigned int n_prts_by_patch[cmprts->n_patches];
  cuda_mparticles_get_size_all(cmprts, n_prts_by_patch);

  //cuda_mparticles_check_in_patch_unordered_slow(cmprts, n_prts_by_patch);

  cuda_mparticles_find_block_indices_ids(cmprts);
  //cuda_mparticles_check_bidx_id_unordered_slow(cmprts, n_prts_by_patch);

  thrust::device_ptr<float4> d_xi4(cmprts->d_xi4);
  thrust::device_ptr<float4> d_pxi4(cmprts->d_pxi4);
  thrust::device_ptr<unsigned int> d_bidx(cmprts->d_bidx);
  thrust::device_ptr<unsigned int> d_id(cmprts->d_id);

  assert(cmprts->n_prts + buf_n <= cmprts->n_alloced);
  thrust::copy(h_xi4.begin(), h_xi4.end(), d_xi4 + cmprts->n_prts);
  thrust::copy(h_pxi4.begin(), h_pxi4.end(), d_pxi4 + cmprts->n_prts);
  thrust::copy(h_bidx.begin(), h_bidx.end(), d_bidx + cmprts->n_prts);
  //thrust::copy(h_id.begin(), h_id.end(), d_id + cmprts->n_prts);
  thrust::sequence(d_id, d_id + cmprts->n_prts + buf_n);

  // for (int i = -5; i <= 5; i++) {
  //   //    float4 xi4 = d_xi4[cmprts->n_prts + i];
  //   unsigned int bidx = d_bidx[cmprts->n_prts + i];
  //   unsigned int id = d_id[cmprts->n_prts + i];
  //   printf("i %d bidx %d %d\n", i, bidx, id);
  // }

  // cuda_mparticles_check_ordered(cmprts);

  cmprts->n_prts += buf_n;

  thrust::stable_sort_by_key(d_bidx, d_bidx + cmprts->n_prts, d_id);
  cuda_mparticles_reorder_and_offsets(cmprts);

  // cuda_mparticles_check_ordered(cmprts);
}

// ----------------------------------------------------------------------
// cuda_mparticles_set_particles

void
cuda_mparticles_set_particles(struct cuda_mparticles *cmprts, unsigned int n_prts, unsigned int off,
			      void (*get_particle)(struct cuda_mparticles_prt *prt, int n, void *ctx),
			      void *ctx)
{
  float4 *xi4  = new float4[n_prts];
  float4 *pxi4 = new float4[n_prts];
  
  for (int n = 0; n < n_prts; n++) {
    struct cuda_mparticles_prt prt;
    get_particle(&prt, n, ctx);

    for (int d = 0; d < 3; d++) {
      int bi = fint(prt.xi[d] * cmprts->b_dxi[d]);
      if (bi < 0 || bi >= cmprts->b_mx[d]) {
	printf("XXX xi %g %g %g\n", prt.xi[0], prt.xi[1], prt.xi[2]);
	printf("XXX n %d d %d xi4[n] %g biy %d // %d\n",
	       n, d, prt.xi[d], bi, cmprts->b_mx[d]);
	if (bi < 0) {
	  prt.xi[d] = 0.f;
	} else {
	  prt.xi[d] *= (1. - 1e-6);
	}
      }
      bi = floorf(prt.xi[d] * cmprts->b_dxi[d]);
      assert(bi >= 0 && bi < cmprts->b_mx[d]);
    }

    xi4[n].x  = prt.xi[0];
    xi4[n].y  = prt.xi[1];
    xi4[n].z  = prt.xi[2];
    xi4[n].w  = cuda_int_as_float(prt.kind);
    pxi4[n].x = prt.pxi[0];
    pxi4[n].y = prt.pxi[1];
    pxi4[n].z = prt.pxi[2];
    pxi4[n].w = prt.qni_wni;
  }

  cuda_mparticles_to_device(cmprts, (float_4 *) xi4, (float_4 *) pxi4, n_prts, off);
  
  delete[] xi4;
  delete[] pxi4;
}

// ----------------------------------------------------------------------
// cuda_mparticles_get_particles

void
cuda_mparticles_get_particles(struct cuda_mparticles *cmprts, unsigned int n_prts, unsigned int off,
			      void (*put_particle)(struct cuda_mparticles_prt *, int, void *),
			      void *ctx)
{
  float4 *xi4  = new float4[n_prts];
  float4 *pxi4 = new float4[n_prts];

  cuda_mparticles_reorder(cmprts);
  cuda_mparticles_from_device(cmprts, (float_4 *) xi4, (float_4 *) pxi4, n_prts, off);
  
  for (int n = 0; n < n_prts; n++) {
    struct cuda_mparticles_prt prt;
    prt.xi[0]   = xi4[n].x;
    prt.xi[1]   = xi4[n].y;
    prt.xi[2]   = xi4[n].z;
    prt.kind    = cuda_float_as_int(xi4[n].w);
    prt.pxi[0]  = pxi4[n].x;
    prt.pxi[1]  = pxi4[n].y;
    prt.pxi[2]  = pxi4[n].z;
    prt.qni_wni = pxi4[n].w;

    put_particle(&prt, n, ctx);

#if 0
    for (int d = 0; d < 3; d++) {
      int bi = fint(prt.xi[d] * cmprts->b_dxi[d]);
      if (bi < 0 || bi >= cmprts->b_mx[d]) {
	MHERE;
	mprintf("XXX xi %.10g %.10g %.10g\n", prt.xi[0], prt.xi[1], prt.xi[2]);
	mprintf("XXX n %d d %d xi %.10g b_dxi %.10g bi %d // %d\n",
		n, d, prt.xi[d] * cmprts->b_dxi[d], cmprts->b_dxi[d], bi, cmprts->b_mx[d]);
      }
    }
#endif
  }

  delete[] (xi4);
  delete[] (pxi4);
}

// ----------------------------------------------------------------------
// cuda_mparticles_patch_get_b_dxi

const particle_cuda_real_t *
cuda_mparticles_patch_get_b_dxi(struct cuda_mparticles *cmprts, int p)
{
  return cmprts->b_dxi;
}

// ----------------------------------------------------------------------
// cuda_mparticles_patch_get_b_mx

const int *
cuda_mparticles_patch_get_b_mx(struct cuda_mparticles *cmprts, int p)
{
  return cmprts->b_mx;
}

