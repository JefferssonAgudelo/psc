#include "hip/hip_runtime.h"

#include "grid.hxx"
#include "fields.hxx"
#include "cuda_mfields.h"
#include "cuda_mparticles.h"
#include "cuda_push_particles.cuh"
#include "push_particles_cuda_impl.hxx"

#include "cuda_test.hxx"

#include "../vpic/PscRng.h"

#include <memory>

#include "gtest/gtest.h"

// Rng hackiness

using Rng = PscRng;
using RngPool = PscRngPool<Rng>;

// enum hackiness

enum { // FIXME, duplicated
#if 0
  JXI, JYI, JZI,
  EX , EY , EZ ,
  HX , HY , HZ ,
#endif
  N_FIELDS = 9,
};

// profile hackiness

#include "mrc_profile.h"

struct prof_globals prof_globals; // FIXME

int
prof_register(const char *name, float simd, int flops, int bytes)
{
  return 0;
}

using CudaMparticles = cuda_mparticles<BS144>;

// ======================================================================
// class PushMprtsTest

struct PushMprtsTest : TestBase<CudaMparticles>, ::testing::Test
{
  std::unique_ptr<Grid_t> grid_;

  RngPool rngpool;
  
  const double L = 1e10;
  const Int3 bs_ = { 1, 1, 1 };

  void SetUp()
  {
    auto domain = Grid_t::Domain{{1, 1, 1}, {L, L, L}};
    grid_.reset(new Grid_t{domain});
  }

  // FIXME, convenient interfaces like make_cmflds, make_cmprts
  // should be available generally
  template<typename S>
  std::unique_ptr<cuda_mfields> make_cmflds(S set)
  {
    auto cmflds = std::unique_ptr<cuda_mfields>(new cuda_mfields(*grid_, N_FIELDS, { 0, 2, 2 }));

    fields_single_t flds = cmflds->get_host_fields();
    Fields3d<fields_single_t> F(flds);

    F(EX, 0,0,0) = set(EX);
    F(EX, 0,1,0) = set(EX);
    F(EX, 0,0,1) = set(EX);
    F(EX, 0,1,1) = set(EX);
    
    F(EY, 0,0,0) = set(EY);
    F(EY, 0,0,1) = set(EY);
    //    F(EY, 1,0,0) = set(EY);
    //    F(EY, 1,0,1) = set(EY);
    
    F(EZ, 0,0,0) = set(EZ);
    //    F(EZ, 1,0,0) = set(EZ);
    F(EZ, 0,1,0) = set(EZ);
    //    F(EZ, 1,1,0) = set(EZ);

    F(HX, 0,0,0) = set(HX);
    F(HX, 1,0,0) = set(HX);

    F(HY, 0,0,0) = set(HY);
    F(HY, 0,1,0) = set(HY);

    F(HZ, 0,0,0) = set(HZ);
    F(HZ, 0,0,1) = set(HZ);

    cmflds->copy_to_device(0, flds, 0, N_FIELDS);
    cmflds->dump("accel.fld.json");
    flds.dtor();
  
    return cmflds;
  }

};

// ======================================================================
// Accel test

TEST_F(PushMprtsTest, Accel)
{
  const int n_prts = 131;
  const int n_steps = 10;
  const CudaMparticles::real_t eps = 1e-6;

  // init fields
  auto cmflds = make_cmflds([&] (int m) -> cuda_mfields::real_t {
      switch(m) {
      case EX: return 1.;
      case EY: return 2.;
      case EZ: return 3.;
      default: return 0.;
      }
    });

  // init particles
  Rng *rng = rngpool[0];

  grid_->kinds.push_back(Grid_t::Kind(1., 1., "test_species"));
  std::unique_ptr<CudaMparticles> cmprts(make_cmprts(*grid_, n_prts, [&](int i) -> cuda_mparticles_prt {
	cuda_mparticles_prt prt = {};
	prt.xi[0] = rng->uniform(0, L);
	prt.xi[1] = rng->uniform(0, L);
	prt.xi[2] = rng->uniform(0, L);
	prt.qni_wni = 1.;
	return prt;
      }));
  
  // run test
  for (int n = 0; n < n_steps; n++) {
    CudaPushParticles_<CudaConfig1vbec3d<dim_yz, BS144>>::push_mprts(cmprts.get(), cmflds.get());

    cmprts->get_particles(0, [&] (int i, const cuda_mparticles_prt &prt) {
	EXPECT_NEAR(prt.pxi[0], 1*(n+1), eps);
	EXPECT_NEAR(prt.pxi[1], 2*(n+1), eps);
	EXPECT_NEAR(prt.pxi[2], 3*(n+1), eps);
      });
  }
}

// ======================================================================
// Cyclo test

TEST_F(PushMprtsTest, Cyclo)
{
  const int n_prts = 131;
  const int n_steps = 64;
  // the errors here are (substantial) truncation error, not
  // finite precision, and they add up
  // (but that's okay, if a reminder that the 6th order Boris would
  //  be good)
  const CudaMparticles::real_t eps = 1e-2;

  // init fields
  auto cmflds = make_cmflds([&] (int m) -> cuda_mfields::real_t {
      switch(m) {
      case HZ: return 2. * M_PI / n_steps;
      default: return 0.;
      }
    });

  // init particles
  Rng *rng = rngpool[0];

  grid_->kinds.push_back(Grid_t::Kind(2., 1., "test_species"));
  std::unique_ptr<CudaMparticles> cmprts(make_cmprts(*grid_, n_prts, [&](int i) -> cuda_mparticles_prt {
	cuda_mparticles_prt prt = {};
	prt.xi[0] = rng->uniform(0, L);
	prt.xi[1] = rng->uniform(0, L);
	prt.xi[2] = rng->uniform(0, L);
	prt.pxi[0] = 1.; // gamma = 2
	prt.pxi[1] = 1.;
	prt.pxi[2] = 1.;
	prt.qni_wni = rng->uniform(0, 1.);;
	return prt;
      }));

  // run test
  for (int n = 0; n < n_steps; n++) {
    CudaPushParticles_<CudaConfig1vbec3d<dim_yz, BS144>>::push_mprts(cmprts.get(), cmflds.get());

    double ux = (cos(2*M_PI*(0.125*n_steps-(n+1))/(double)n_steps) /
		 cos(2*M_PI*(0.125*n_steps)      /(double)n_steps));
    double uy = (sin(2*M_PI*(0.125*n_steps-(n+1))/(double)n_steps) /
		 sin(2*M_PI*(0.125*n_steps)      /(double)n_steps));
    double uz = 1.;
    cmprts->get_particles(0, [&] (int i, const cuda_mparticles_prt &prt) {
	EXPECT_NEAR(prt.pxi[0], ux, eps);
	EXPECT_NEAR(prt.pxi[1], uy, eps);
	EXPECT_NEAR(prt.pxi[2], uz, eps);
      });
  }
}

